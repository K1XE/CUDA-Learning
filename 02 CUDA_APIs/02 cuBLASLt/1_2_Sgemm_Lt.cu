#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <iostream>
#include <vector>

// Dimensions m and k must be multiples of 4.
#define M 4
#define K 4
#define N 4

#undef PRINT_MATRIX
#define PRINT_MATRIX(mat, rows, cols)              \
    for (int i = 0; i < rows; i++)                 \
    {                                              \
        for (int j = 0; j < cols; j++)             \
        {                                          \
            std::cout << mat[i * cols + j] << " "; \
        }                                          \
        std::cout << std::endl;                    \
    }

#define CHECK_CUDA(val)                                     \
    do                                                      \
    {                                                       \
        hipError_t sta = val;                              \
        if (sta != hipSuccess)                             \
        {                                                   \
            std::cerr << "CUDA异常，具体信息：" << __FILE__ \
                      << " " << __LINE__                    \
                      << " " << hipGetErrorString(sta)     \
                      << std::endl;                         \
            exit(EXIT_FAILURE);                             \
        }                                                   \
    } while (0)

#define CHECK_CUBLAS(val)                                     \
    do                                                        \
    {                                                         \
        hipblasStatus_t sta = val;                             \
        if (sta != HIPBLAS_STATUS_SUCCESS)                     \
        {                                                     \
            std::cerr << "CUBLAS异常，具体信息：" << __FILE__ \
                      << " " << __LINE__                      \
                      << " " << sta                           \
                      << std::endl;                           \
            exit(EXIT_FAILURE);                               \
        }                                                     \
    } while (0)

void cpuMatMul(float *A, float *B, float *C)
{
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            float sums = 0.0f;
            for (int l = 0; l < K; l++)
            {
                sums += A[i * K + l] * B[l * N + j];
            }
            C[i * N + j] = sums;
        }
    }
}

int main(int argc, char **argv)
{
    float h_A[M * K] = {
        1.0f, 2.0f, 3.0f, 4.0f,
        5.0f, 6.0f, 7.0f, 8.0f,
        9.0f, 10.0f, 11.0f, 12.0f,
        13.0f, 14.0f, 15.0f, 16.0f};
    float h_B[K * N] = {
        1.0f, 2.0f, 4.0f, 4.0f,
        7.0f, 7.0f, 7.0f, 7.0f,
        9.0f, 10.0f, 11.0f, 12.0f,
        13.0f, 14.0f, 15.0f, 16.0f};
    float h_C_cpu[M * N] = {0};
    float h_C_gpu[M * N] = {0};

    float *d_A, *d_B, *d_C;

    CHECK_CUDA(hipMalloc(&d_A, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, M * N * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));

    hipblasLtHandle_t handleLt1;
    hipblasLtCreate(&handleLt1);

    hipblasLtMatrixLayout_t matA_fp32, matB_fp32, matC_fp32;
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matA_fp32, HIP_R_32F, K, M, K));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matB_fp32, HIP_R_32F, N, K, N));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matC_fp32, HIP_R_32F, N, M, N));

    hipblasLtMatmulDesc_t matmulDesc_fp32;
    CHECK_CUBLAS(hipblasLtMatmulDescCreate(&matmulDesc_fp32, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp32, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(hipblasOperation_t)));
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp32, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(hipblasOperation_t)));

    float alpha = 1.0f;
    float beta = 0.0f;

    CHECK_CUBLAS(hipblasLtMatmul(
        handleLt1,
        matmulDesc_fp32,
        &alpha,
        d_B, matB_fp32,
        d_A, matA_fp32,
        &beta,
        d_C, matC_fp32,
        d_C, matC_fp32,
        NULL, NULL,
        0, 0
    ));

    CHECK_CUDA(hipMemcpy(h_C_gpu, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));
    
    cpuMatMul(h_A, h_B, h_C_cpu);

    std::cout << "矩阵 A 为（" << M << "x" << K << "）：" << std::endl;
    PRINT_MATRIX(h_A, M, K);
    std::cout << "矩阵 B 为（" << K << "x" << N << "）：" << std::endl;
    PRINT_MATRIX(h_B, K, N);

    std::cout << "CPU 运算结果（" << M << "x" << N << "）：" << std::endl;
    PRINT_MATRIX(h_C_cpu, M, N);
    std::cout << "cuBLASLt 半精度通用矩阵乘法结果（" << M << "x" << N << "）：" << std::endl;
    PRINT_MATRIX(h_C_gpu, M, N);


    // 正确性测试
    bool correct_fp32 = 1;
    for (int i = 0; i < M * N; i ++ )
    {
        if (fabs(h_C_cpu[i] - h_C_gpu[i]) > 1e-5f)
        {
            correct_fp32 = 0;
            exit(EXIT_FAILURE);
        }
    }
    if (correct_fp32) std::cout << "运算误差在容忍度内！" << std::endl;

    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matA_fp32));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matB_fp32));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matC_fp32));
    CHECK_CUBLAS(hipblasLtMatmulDescDestroy(matmulDesc_fp32));
    CHECK_CUBLAS(hipblasLtDestroy(handleLt1));

    return 0;
}